#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

#define RENDER_NORMALS 0
#define RENDER_DEPTH 0
#define RENDER_TEXTURE 1

#define LAMBERT 0
#define BILINEAR_FILTER 1

#define RENDER_POINT 0
#define RENDER_LINE 0
#define RENDER_TRIANGLE 1

#define BACKFACE_CULLING 1

PerformanceTimer& timer()
{
	static PerformanceTimer timer;
	return timer;
}

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 VertexAttributeTexcoord texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		 int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;
		 VertexAttributeTexcoord texcoord0;
		 TextureData* dev_diffuseTex;
		 int texWidth, texHeight;
		 float z;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
		glm::vec3 color = glm::vec3(0.0f, 0.0f, 0.0f);
		Fragment fragment = fragmentBuffer[index];

		// TODO: add your fragment shader code here
#if RENDER_TEXTURE
		glm::vec3 diffuseCol;
		if (fragment.dev_diffuseTex != NULL) {
			TextureData *texture = fragment.dev_diffuseTex;

	#if BILINEAR_FILTER
			glm::vec2 UV = glm::vec2(fragment.texcoord0.x * fragment.texWidth, fragment.texcoord0.y * fragment.texHeight);
			glm::ivec2 UV_Int = glm::ivec2((int)UV.x, (int)UV.y);

			float delta_x = UV.x - UV_Int.x;
			float delta_y = UV.y - UV_Int.y;

			int colIdx_01 = UV_Int.x + UV_Int.y * fragment.texWidth;
			int colIdx_02 = UV_Int.x + 1 + UV_Int.y * fragment.texWidth;
			int colIdx_03 = UV_Int.x + (UV_Int.y + 1) * fragment.texWidth;
			int colIdx_04 = UV_Int.x + 1 + (UV_Int.y + 1) * fragment.texWidth;
			
			glm::vec3 diffuse_01 = glm::vec3(texture[colIdx_01 * 3], texture[colIdx_01 * 3 + 1], texture[colIdx_01 * 3 + 2]);
			glm::vec3 diffuse_02 = glm::vec3(texture[colIdx_02 * 3], texture[colIdx_02 * 3 + 1], texture[colIdx_02 * 3 + 2]);
			glm::vec3 diffuse_03 = glm::vec3(texture[colIdx_03 * 3], texture[colIdx_03 * 3 + 1], texture[colIdx_03 * 3 + 2]);
			glm::vec3 diffuse_04 = glm::vec3(texture[colIdx_04 * 3], texture[colIdx_04 * 3 + 1], texture[colIdx_04 * 3 + 2]);

			glm::vec3 diffuse_interp_1 = glm::mix(diffuse_01, diffuse_02, delta_x);
			glm::vec3 diffuse_interp_2 = glm::mix(diffuse_03, diffuse_04, delta_x);
			diffuseCol = glm::mix(diffuse_interp_1, diffuse_interp_2, delta_y);
			diffuseCol /= 255.0f;


	#else 
			glm::ivec2 UV = glm::ivec2((int)(fragment.texcoord0.x * fragment.texWidth), (int)(fragment.texcoord0.y * fragment.texHeight));
			int colIdx = UV.x + UV.y * fragment.texWidth;
			diffuseCol.r = texture[colIdx * 3];
			diffuseCol.g = texture[colIdx * 3 + 1];
			diffuseCol.b = texture[colIdx * 3 + 2];
			diffuseCol /= 255.0f;
	#endif

		}
		else {
			diffuseCol = fragment.color;
		}

	#if LAMBERT
			glm::vec3 lightPos = glm::vec3(10.0f, 10.0f, 10.0f);
			glm::vec3 lightVec = glm::normalize(lightPos - fragment.eyePos); // point light
			//glm::vec3 lightVec = glm::vec3(1.0f, 1.0f, 1.0f);  // directional light
			float lambert_term = glm::dot(lightVec, fragment.eyeNor);
			color = diffuseCol * lambert_term;
	#else
			color = diffuseCol;
	#endif


#endif

#if RENDER_POINT || RENDER_LINE
			color = fragment.color;
#endif

#if RENDER_NORMALS 
		color = glm::abs(fragment.eyeNor);
		//color.r = (fragment.eyeNor.x + 1.0f) / 2.0f;
		//color.g = (fragment.eyeNor.y + 1.0f) / 2.0f;
		//color.b = (fragment.eyeNor.z + 1.0f) / 2.0f;

#endif

#if RENDER_DEPTH
		float z = fragment.z;
		z = (z + 1.0f) / 2.0f;
		color = glm::vec3(z, z, z);
		//color = glm::vec3(1.0f - depth, 1.0f - depth, 1.0f - depth);  // depth as color
#endif



		framebuffer[index] = color;
		

    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		glm::vec4 pos = MVP * glm::vec4(primitive.dev_position[vid], 1.0f);
		glm::vec4 eyePos = MV * glm::vec4(primitive.dev_position[vid], 1.0f);
		// Then divide the pos by its w element to transform into NDC space
		pos /= pos.w;
		// Finally transform x and y to viewport space
		pos.x = (pos.x + 1.0f) * 0.5f * width;
		pos.y = (1.0f - pos.y) * 0.5f * height;
		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array
		primitive.dev_verticesOut[vid].pos = pos;
		primitive.dev_verticesOut[vid].eyePos = glm::vec3(eyePos);
		primitive.dev_verticesOut[vid].eyeNor = glm::normalize(MV_normal * primitive.dev_normal[vid]);

		if (primitive.dev_diffuseTex != NULL) {
			primitive.dev_verticesOut[vid].dev_diffuseTex = primitive.dev_diffuseTex;
			primitive.dev_verticesOut[vid].texWidth = primitive.diffuseTexWidth;
			primitive.dev_verticesOut[vid].texHeight = primitive.diffuseTexHeight;
			primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];
		}
		

	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}


		// TODO: other primitive types (point, line)
	}
	
}

__device__
void rasterizePoint(glm::vec4 p, glm::vec3 color, Fragment *fragments, int width, int height) {
	int x = (int)p.x;
	int y = (int)p.y;

	x = x >= width ? width - 1 : x;
	x = x < 0 ? 0 : x;
	y = y >= height ? height - 1 : y;
	y = y < 0 ? 0 : y;

	int idx = x + y * width;
	fragments[idx].color = color;

}

__device__
void rasterizeLine(glm::vec4 p1, glm::vec4 p2, glm::vec3 color, Fragment *fragments, int width, int height) {
	int x1 = glm::clamp(p1.x, 0.f, (float)(width - 1));
	int x2 = glm::clamp(p2.x, 0.f, (float)(width - 1));
	int y1 = glm::clamp(p1.y, 0.f, (float)(height - 1));
	int y2 = glm::clamp(p2.y, 0.f, (float)(height - 1));

	if (x1 == x2) {
		for (int y = y1; y < y2; ++y) {
			int idx = x1 + y * width;
			fragments[idx].color = color;
		}
	} else {
		int delta_x = x2 - x1;
		int delta_y = y2 - y1;
		for (int x = x1; x < x2; ++x) {
			float y = y1 + delta_y * (x - x1) / delta_x;
			int idx = x + (int)y * width;
			fragments[idx].color = color;
		}
	}
	

}

__device__
void rasterizeTriangle(glm::vec3 tri[3], glm::vec3 color, Primitive primitive, Fragment *fragments, int* depths, int width, int height) {
	AABB boundingBox = getAABBForTriangle(tri);
	if (boundingBox.min.x >= width || boundingBox.min.y >= height || boundingBox.max.x < 0 || boundingBox.max.y < 0) {
		return;
	}
	for (int y = boundingBox.min.y; y < boundingBox.max.y; y++) {
		for (int x = boundingBox.min.x; x < boundingBox.max.x; x++) {
			if (y >= height || y < 0 || x >= width || x < 0) {
				continue;
			}
			int idx = x + y * width;
			glm::vec3 bc = calculateBarycentricCoordinate(tri, glm::vec2(x, y));
			if (isBarycentricCoordInBounds(bc)) {
				float z = getZAtCoordinate(bc, tri);
				int depth = (int)-100 * z;
				atomicMin(&depths[idx], depth);
				if (depth == depths[idx]) {
					fragments[idx].z = z;

					fragments[idx].color = color;  // plain white color

					fragments[idx].dev_diffuseTex = primitive.v[0].dev_diffuseTex;
					fragments[idx].texHeight = primitive.v[0].texHeight;
					fragments[idx].texWidth = primitive.v[0].texWidth;

					glm::vec3 eyePostions[3];
					eyePostions[0] = primitive.v[0].eyePos;
					eyePostions[1] = primitive.v[1].eyePos;
					eyePostions[2] = primitive.v[2].eyePos;

					glm::vec3 eyeNormals[3];
					eyeNormals[0] = primitive.v[0].eyeNor;
					eyeNormals[1] = primitive.v[1].eyeNor;
					eyeNormals[2] = primitive.v[2].eyeNor;
					fragments[idx].eyePos = BCInterpVector(bc, eyePostions);
					fragments[idx].eyeNor = BCInterpVector(bc, eyeNormals);

					glm::vec2 UVs[3];
					UVs[0] = primitive.v[0].texcoord0;
					UVs[1] = primitive.v[1].texcoord0;
					UVs[2] = primitive.v[2].texcoord0;
					fragments[idx].texcoord0 = PCInterpUV(bc, eyePostions, UVs);

				}
			}
		}
	}
}

__global__
void rasterizePrimitives(int totalNumPrimitives, Primitive *primitives, Fragment *fragments, int* depths, int width, int height) {
	int pid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (pid >= totalNumPrimitives) {
		return;
	}

	glm::vec3 color = glm::vec3(1.0f, 1.0f, 1.0f);
	Primitive primitive = primitives[pid];
	glm::vec3 tri[3];
	tri[0] = glm::vec3(primitive.v[0].pos);
	tri[1] = glm::vec3(primitive.v[1].pos);
	tri[2] = glm::vec3(primitive.v[2].pos);

#if BACKFACE_CULLING
	if (isBackface(tri)) {
		return;
	}
#endif

#if RENDER_POINT
	rasterizePoint(primitive.v[0].pos, color, fragments, width, height);
	rasterizePoint(primitive.v[1].pos, color, fragments, width, height);
	rasterizePoint(primitive.v[2].pos, color, fragments, width, height);


#elif RENDER_LINE
	rasterizeLine(primitive.v[0].pos, primitive.v[1].pos, color, fragments, width, height);
	rasterizeLine(primitive.v[1].pos, primitive.v[2].pos, color, fragments, width, height);
	rasterizeLine(primitive.v[0].pos, primitive.v[2].pos, color, fragments, width, height);

#elif RENDER_TRIANGLE
	rasterizeTriangle(tri, color, primitive, fragments, depths, width, height);
	
#endif
	
}


/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);

				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	dim3 blockSize(128);
	dim3 numBlocksForPrimitives((totalNumPrimitives + blockSize.x - 1) / blockSize.x);
	rasterizePrimitives << <numBlocksForPrimitives, blockSize >> > (totalNumPrimitives, dev_primitives, dev_fragmentBuffer, dev_depth, width, height);
	checkCUDAError("rasterizer");

    // Copy depthbuffer colors into framebuffer
	timer().startGpuTimer();
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
	timer().endGpuTimer();

	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    checkCUDAError("rasterize Free");
}
