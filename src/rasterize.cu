#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType
	{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut 
	{
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		// int texWidth, texHeight;
		// ...
	};

	struct Primitive 
	{
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		// glm::vec3 eyePos;	// eye space position used for shading
		// glm::vec3 eyeNor;
		// VertexAttributeTexcoord texcoord0;
		// TextureData* dev_diffuseTex;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
		framebuffer[index] = fragmentBuffer[index].color;

		// TODO: add your fragment shader code here

    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

__device__ __host__
glm::vec4 NDCToScreenSpace(glm::vec4* v, int width, int height)
{
	glm::vec4 screenCoords(0.f, 0.f, 0.f, 1.f);
	screenCoords[0] = ((*v)[0] + 1.f) * width / 2.f;
	screenCoords[1] = (1.f - (*v)[1]) * height / 2.f;
	return screenCoords;
}

__device__ __host__
glm::vec4 GetTriangleBounds(glm::vec4* v1, glm::vec4* v2, glm::vec4* v3, int width, int height)
{
	glm::vec4 bounds(0.f, 0.f, 0.f, 1.f);

	const float minX = glm::min(glm::min((*v1)[0], (*v2)[0]), (*v3)[0]);
	const float maxX = glm::max(glm::max((*v1)[0], (*v2)[0]), (*v3)[0]);
	
	const float minY = glm::min(glm::min((*v1)[1], (*v2)[1]), (*v3)[1]);
	const float maxY = glm::max(glm::max((*v1)[1], (*v2)[1]), (*v3)[1]);

	bounds[0] = glm::clamp(minX, 0.f, float(width));
	bounds[1] = glm::clamp(minY, 0.f, float(height));
	bounds[2] = glm::clamp(maxX, 0.f, float(width));
	bounds[3] = glm::clamp(maxY, 0.f, float(height));

	return bounds;
}

__host__ __device__
glm::vec3 GetBaryCentric(const glm::vec3* currentPoint, const glm::vec3* p1, const glm::vec3* p2, const glm::vec3* p3)
{
	const glm::vec3 s1 = glm::cross(((*currentPoint) - (*p2)), ((*p3) - (*p2)));
	const glm::vec3 s2 = glm::cross(((*currentPoint) - (*p1)), ((*p2) - (*p1)));
	const glm::vec3 s3 = glm::cross(((*currentPoint) - (*p3)), ((*p1) - (*p3)));
	const float area = glm::length(glm::cross(((*p1) - (*p2)) , ((*p3) - (*p2))));
	const glm::vec3 influence = glm::vec3(0.f);// glm::vec3(glm::length(s1) / area, glm::length(s2) / area, glm::length(s3) / area);
	return influence;
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}
}

//------------------------- SHADERS -----------------------------------------------------------------

static int curPrimitiveBeginId = 0;

__global__ 
void _vertexTransformAndAssembly(
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int numVertices = primitive.numVertices;

	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space

		for (int i = 0; i < numVertices; ++i)
		{
			const VertexAttributePosition inPos = primitive.dev_position[i];
			
			// This is in NDC Space
			glm::vec4 outPos = MVP * glm::vec4(inPos, 1.0f);

			// Convert to screen Space
			primitive.dev_verticesOut[i].pos = NDCToScreenSpace(&outPos, width, height);
		}

		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array
		
	}
}

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;

			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]	= primitive.dev_verticesOut[primitive.dev_indices[iid]];
			dev_primitives[pid + curPrimitiveBeginId].primitiveType = primitive.primitiveType;


		}
		// TODO: other primitive types (point, line)
	}
}


__global__
void _rasterizePrimitive(int width, int height, int totalNumPrimitives, Primitive* dev_primitives, Fragment* dev_fragmentBuffer, int* dev_depth)
{
	const int primitiveId = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (primitiveId < totalNumPrimitives)
	{
		Primitive& primitive = dev_primitives[primitiveId];

		if (primitive.primitiveType == Triangle) 
		{
			// Vertices in screen Space
			VertexOut v1 = primitive.v[0];
			VertexOut v2 = primitive.v[1];
			VertexOut v3 = primitive.v[2];

			glm::vec3 triangle[3];
			triangle[0] = glm::vec3(v1.pos);
			triangle[1] = glm::vec3(v2.pos);
			triangle[2] = glm::vec3(v3.pos);

			const AABB bounds = getAABBForTriangle(triangle);

			for (int row = bounds.min[1]; row < bounds.max[1]; ++row)
			{
				for (int col = bounds.min[0]; col < bounds.max[0]; ++col)
				{
					const int pixelIndex = col + row * width;
					const glm::vec2 currPos(col, row);
					
					// Calculate BaryCentric coordinates
					const glm::vec3 baryCoord = calculateBarycentricCoordinate(triangle, currPos);

					// Check if point is inside triangle
					const bool isInside = isBarycentricCoordInBounds(baryCoord);

					if (isInside)
					{
						// Get the interop depth
						const float currDepth = getZAtCoordinate(baryCoord, triangle);
						if (currDepth < dev_depth[pixelIndex])
						{
							// Update frame buffer and depth buffer
							dev_fragmentBuffer[pixelIndex].color = glm::vec3(1.0f, 1.0f, 1.0f);
							dev_depth[pixelIndex] = currDepth;
						}
					}
				}
			}
		}
	}
}

__global__
void _fragmentShader()
{

}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				// 1. Vertex Assembly and Shader
				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(*p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");

				hipDeviceSynchronize();

				// 2. Primitive Assembly
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));

	// 3. Depth Check
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// 4. Rasterize - Call per primitive
	{
		dim3 numThreadsPerBlock(128);
		dim3 numBlocksForPrimitives((totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

		_rasterizePrimitive << <numBlocksForPrimitives, numThreadsPerBlock >> > (width, height, totalNumPrimitives, dev_primitives, dev_fragmentBuffer, dev_depth);
		checkCUDAError("Rasterizer");
	}

	// 5. Fragment Shader - call per primitive

    // Copy fragmentBuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");

    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    checkCUDAError("rasterize Free");
}
