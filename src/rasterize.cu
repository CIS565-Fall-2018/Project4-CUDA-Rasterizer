#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <chrono>

#define LAMBERT 1
#define BLINN 0

#define POINT 0
#define TRI 0
#define LINE 1

#define TEXTURE 1
#define BILINEAR 1

#define PERSP_CORRECT 1

#define BACKCULL 0

#define TIMER 0
namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		 glm::vec3 camPos;
		 int texWidth, texHeight;
		 glm::vec3 mvpPos;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;
		 glm::vec3 camPos;
		 VertexAttributeTexcoord texcoord0;
		 TextureData* dev_diffuseTex;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static float * dev_depth = NULL;	// you might need this buffer when doing depth test
static int * dev_mutex = NULL;
/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        

		// TODO: add your fragment shader code here
		Fragment fragment = fragmentBuffer[index];
#if POINT || LINE 
		framebuffer[index] = fragment.color;
#elif LAMBERT
		glm::vec3 v = fragment.eyePos;
		glm::vec3 n = fragment.eyeNor;
		glm::vec3 fragColor(1, 0, 0);
		glm::vec3 lightPos = glm::vec3(2, 2, 2) + v;
		glm::vec3 L = glm::normalize(lightPos - v);
		float lambert = glm::max(0.f, glm::dot(L, n));
		glm::vec3 ambient = glm::vec3(0.1) * fragment.color;
		framebuffer[index] = ambient + 0.9f * lambert * fragment.color;
#elif BLINN
		glm::vec3 lights[2] = { glm::vec3(2, 2, 2) };
		framebuffer[index] = glm::vec3(0.0f);
		for (glm::vec3 lightPos : lights) {
			glm::vec3 v = fragment.eyePos;
			glm::vec3 n = glm::normalize(fragment.eyeNor);
			glm::vec3 camPos = fragment.camPos;
			glm::vec3 L = lightPos;
			glm::vec3 Ev = glm::normalize(-v);
			glm::vec3 R = glm::normalize(-glm::reflect(v - camPos, n));
			glm::vec3 ambient = glm::vec3(0.1) * fragment.color;
			float specular = glm::pow(glm::max(glm::dot(R, Ev), 0.f), 32.f);
			float lambert = glm::max(0.f, glm::dot(L, n));
			glm::vec3 diffuse = lambert * fragment.color;
			framebuffer[index] += glm::clamp(ambient + diffuse*glm::vec3(0.7)
				+ specular * glm::vec3(0.2), glm::vec3(0), glm::vec3(1));
		}
		
#else
		framebuffer[index] = fragment.color;
#endif
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(float));

	hipFree(dev_mutex);
	hipMalloc(&dev_mutex, width * height * sizeof(int));
	hipMemset(dev_mutex, 0, width * height * sizeof(int));
	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, float * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}



/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}


__forceinline__ __device__ glm::vec3 baryPos(VertexOut *v, glm::vec3  barycentric) {
	VertexOut vert0, vert1, vert2;
	vert0 = v[0];
	vert1 = v[1];
	vert2 = v[2];
	glm::vec3 p0 = vert0.mvpPos * barycentric[0];
	glm::vec3 p1 = vert1.mvpPos * barycentric[1];
	glm::vec3 p2 = vert2.mvpPos * barycentric[2];
	return p0 + p1 + p2;			  
}

__forceinline__ __device__ glm::vec3 baryNorm(VertexOut *v, glm::vec3  barycentric) {
	VertexOut vert0, vert1, vert2;
	vert0 = v[0];
	vert1 = v[1];
	vert2 = v[2];
	glm::vec3 p0 = vert0.eyeNor * barycentric[0];
	glm::vec3 p1 = vert1.eyeNor * barycentric[1];
	glm::vec3 p2 = vert2.eyeNor * barycentric[2];
	return p0 + p1 + p2;
}


__forceinline__ __device__ glm::vec2 baryUVs(VertexOut *v, glm::vec3  barycentric) {
	VertexOut vert0, vert1, vert2;
	vert0 = v[0];
	vert1 = v[1];
	vert2 = v[2];
	glm::vec2 p0 = vert0.texcoord0 * barycentric[0];
	glm::vec2 p1 = vert1.texcoord0 * barycentric[1];
	glm::vec2 p2 = vert2.texcoord0 * barycentric[2];
	return p0 + p1 + p2;
}


__forceinline__ __device__ glm::vec2 baryUVsPerspective(VertexOut *v, glm::vec3  barycentric) {
	VertexOut v0, v1, v2;
	v0 = v[0];
	v1 = v[1];
	v2 = v[2];

	glm::vec2 texCoordZ = barycentric.x * (v0.texcoord0 / v0.eyePos.z) + barycentric.y * (v1.texcoord0 / v1.eyePos.z) + barycentric.z * (v2.texcoord0 / v2.eyePos.z);
	float coordZ = barycentric.x * (1.0f / v0.eyePos.z) + barycentric.y * (1.0f / v1.eyePos.z) + barycentric.z * (1.0f / v2.eyePos.z);
	return texCoordZ / coordZ;
}

__device__
glm::vec3 getColor(TextureData* tex, int width, float x, float y) {
	int i = x + y * width;
	return glm::vec3(tex[i * 3], tex[i * 3 + 1], tex[i * 3 + 2]) / 255.f;
}

__global__ void kernelRasterize(int totalNumPrimitives, Primitive *dev_primitives, Fragment *dev_fragmentBuffer, float *dev_depth, int *dev_mutex, int width, int height) {
	int pid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (pid > totalNumPrimitives) return;
	// compute triangle from primitive
	Primitive primitive = dev_primitives[pid];
	glm::vec3 v0, v1, v2;
	v0 = glm::vec3(primitive.v[0].pos);
	v1 = glm::vec3(primitive.v[1].pos);
	v2 = glm::vec3(primitive.v[2].pos);
	glm::vec3 triangle[3] = { v0, v1, v2 };
#if BACKCULL
	if (glm::dot(primitive.v->eyeNor, primitive.v->camPos - primitive.v->eyePos) < -1.f) {
		return;
	}
#endif

#if POINT
int x, y;
for (int i = 0; i < 3; i++) {
	x = triangle[i].x;
	y = triangle[i].y;
	int fragmentId = x + y * width;
	if ((x >= 0 && x <= width - 1) && (y >= 0 && y <= height - 1)) {
		dev_fragmentBuffer[fragmentId].color = primitive.v->eyeNor;
	}
}
#elif LINE
for (int i = 0; i < 3; i++) {
	int x1 = triangle[i].x;
	int x2 = triangle[i + 1].x;
	int y1 = triangle[i].y;
	int y2 = triangle[i + 1].y;
	int dx = x2 - x1;
	int dy = y2 - y1;
	for (int x = x1; x <= x2; x++) {
		int y = y1 + dy * (x - x1) / dx;
		int fragmentId = x + y * width;
		if (x < 0 || x >= width) continue;
		if (y < 0 || y >= height) continue;
		dev_fragmentBuffer[fragmentId].color = primitive.v->eyeNor;
	}
}
#elif TRI
	// compute bounding box and clip to screen
	AABB boundingBox = getAABBForTriangle(triangle);
	const int minX = glm::min(width - 1, glm::max(0, (int)boundingBox.min.x));
	const int minY = glm::min(height - 1, glm::max(0, (int)boundingBox.min.y));
	const int maxX = glm::min(width - 1, glm::max(0, (int)boundingBox.max.x));
	const int maxY = glm::min(height - 1, glm::max(0, (int)boundingBox.max.y));

	// iterate over bounding box and test which pixels are inside
	for (int x = minX; x <= maxX; ++x) {
		for (int y = minY; y <= maxY; ++y) {
			glm::vec3 barycentric = calculateBarycentricCoordinate(triangle, glm::vec2(x, y));
			bool inTriangle = isBarycentricCoordInBounds(barycentric);

			if (inTriangle) {
				const int fragmentId = x + (y * width);

				bool isSet;
				do {
					// it was unlocked so we lock it
					isSet = atomicCAS(&dev_mutex[fragmentId], 0, 1) == 0;
					if (isSet) {
						float depth = -getZAtCoordinate(barycentric, triangle) * INT_MAX;

						// if this fragment is closer, we set the new depth and fragment
						if (depth < dev_depth[fragmentId]) {
							dev_depth[fragmentId] = depth;
							Fragment &fragment = dev_fragmentBuffer[fragmentId];
							fragment.eyeNor = baryNorm(primitive.v, barycentric);
							fragment.eyePos = baryPos(primitive.v, barycentric);
							fragment.color = fragment.eyeNor;
							fragment.color = glm::vec3(1.f);
							fragment.camPos = primitive.v[0].camPos;
#if PERSP_CORRECT
							glm::vec2 uvs = baryUVsPerspective(primitive.v, barycentric);
							fragment.texcoord0 = uvs;
							auto v0 = primitive.v[0];
							auto v1 = primitive.v[1];
							auto v2 = primitive.v[2];
							glm::vec2 texCoordZ = barycentric.x * (v0.texcoord0 / v0.eyePos.z) + barycentric.y * (v1.texcoord0 / v1.eyePos.z) + barycentric.z * (v2.texcoord0 / v2.eyePos.z);
							float coordZ = barycentric.x * (1.0f / v0.eyePos.z) + barycentric.y * (1.0f / v1.eyePos.z) + barycentric.z * (1.0f / v2.eyePos.z);
							fragment.texcoord0 = texCoordZ / coordZ;
#else
							glm::vec2 uvs = baryUVs(primitive.v, barycentric);
							fragment.texcoord0 = uvs;
#endif 

#if BILINEAR
							if (primitive.v->dev_diffuseTex != NULL) {
								float x = uvs[0] * primitive.v->texWidth;
								float y = uvs[1] * primitive.v->texHeight;
								int xx = glm::floor(x);
								int yy = glm::floor(y);
								float xfract = x - xx;
								float yfract = y - yy;
								float xinv = 1.f - xfract;
								float yinv = 1.f - yfract;
							
								TextureData *text = primitive.v->dev_diffuseTex;
								int width = primitive.v->texWidth;
								glm::vec3 tex00 = getColor(text, width, xx, yy);
								glm::vec3 tex10 = getColor(text, width, xx + 1, yy);
								glm::vec3 tex01 = getColor(text, width, xx, yy + 1);
								glm::vec3 tex11 = getColor(text, width, xx + 1, yy + 1);

								fragment.color = (tex00 * xinv + tex10 * xfract) * yinv + (tex01 * xinv + tex11 * xfract) * yfract;
							}
#elif TEXTURE
							if (primitive.v->dev_diffuseTex != NULL) {
								float x = fragment.texcoord0[0] * primitive.v->texWidth;
								float y = fragment.texcoord0[1] * primitive.v->texHeight;
								TextureData *text = primitive.v->dev_diffuseTex;
								int width = primitive.v->texWidth;
								fragment.color = getColor(text, width, glm::floor(x), glm::floor(y));
							}
#else
							//fragment.color = baryNorm(primitive.v, barycentric);
#endif
						}
						dev_mutex[fragmentId] = 0;
					}
				} while (!isSet);
			}
		}
	}
#endif
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}

	

	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space
		glm::vec4 objPos(primitive.dev_position[vid], 1.f);
		glm::vec4 eyePos = MVP * objPos;
		glm::vec4 mvpPos = eyePos;
		mvpPos /= mvpPos.w;
		mvpPos.x = 0.5f * float(width) * (mvpPos.x + 1.f);
		mvpPos.y = 0.5f * float(height) * (1.f - mvpPos.y);
		mvpPos.z = -mvpPos.z;

		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array
		VertexOut &vo = primitive.dev_verticesOut[vid];
		vo.pos = mvpPos;

		vo.eyePos = glm::vec3(MV * objPos);//glm::vec3(eyePos[0], eyePos[1], eyePos[2]);
		vo.mvpPos = glm::vec3(MVP * objPos);
		vo.eyeNor = glm::normalize(MV_normal * primitive.dev_normal[vid]);
		vo.camPos = glm::vec3(MV * glm::vec4(0, 0, 0, 1));
		if (primitive.dev_texcoord0) vo.texcoord0 = primitive.dev_texcoord0[vid];
		if (primitive.dev_diffuseTex) vo.dev_diffuseTex = primitive.dev_diffuseTex;
		vo.texHeight = primitive.diffuseTexHeight;
		vo.texWidth = primitive.diffuseTexWidth;
	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}


		// TODO: other primitive types (point, line)
	}
	
}



/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
	using time_point_t = std::chrono::high_resolution_clock::time_point;
	time_point_t start_time, end_time;
	float elapsed_time;
	std::chrono::duration<double, std::milli> dur;

    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();
		

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				

#if TIMER
				start_time = std::chrono::high_resolution_clock::now();
#endif
				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
#if TIMER
				hipDeviceSynchronize();
				end_time = std::chrono::high_resolution_clock::now();
				dur = end_time - start_time;
				elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
				std::cout << "vertex processing elapsed time: " << elapsed_time << "ms." << std::endl;
#endif

#if TIMER
				start_time = std::chrono::high_resolution_clock::now();
#endif
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
#if TIMER
				hipDeviceSynchronize();
				end_time = std::chrono::high_resolution_clock::now();
				dur = end_time - start_time;
				elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
				std::cout << "primitive assembly elapsed time: " << elapsed_time << "ms." << std::endl;
#endif
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	dim3 numThreadsPerBlock(128);
	dim3 numBlocksForPrimitives((totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
#if TIMER
	start_time = std::chrono::high_resolution_clock::now();
#endif
	kernelRasterize << <numBlocksForPrimitives, numThreadsPerBlock >> > (totalNumPrimitives, dev_primitives, dev_fragmentBuffer, dev_depth, dev_mutex, width, height);
#if TIMER
	hipDeviceSynchronize();
	end_time = std::chrono::high_resolution_clock::now();
	dur = end_time - start_time;
	elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
	std::cout << "rasterize elapsed time: " << elapsed_time << "ms." << std::endl;
#endif

    // Copy depthbuffer colors into framebuffer
#if TIMER
	start_time = std::chrono::high_resolution_clock::now();
#endif
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
#if TIMER
	hipDeviceSynchronize();
	end_time = std::chrono::high_resolution_clock::now();
	dur = end_time - start_time;
	elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
	std::cout << "fragment shader elapsed time: " << elapsed_time << "ms." << std::endl;
#endif

	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    checkCUDAError("rasterize Free");
}
