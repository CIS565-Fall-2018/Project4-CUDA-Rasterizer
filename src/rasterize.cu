#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <>

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType {
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		glm::vec3 color;
		glm::vec2 texcoord0;
		TextureData* dev_diffuseTex = NULL;
		int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static float * dev_depth = NULL;	// you might need this buffer when doing depth test

static int *mutex;

static int width = 0;
static int height = 0;
static int originalWidth = 0;
static int originalHeight = 0;
static int antialiasing = 2;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image, int aaSize, int largeW) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		glm::vec3 color;
		color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
		color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
		color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

/**
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	const int numLights = 2;
	glm::vec3 lights[numLights] = { glm::normalize(glm::vec3(-1,-1,-1)), glm::normalize(glm::vec3(1,1,1)) };

	if (x < w && y < h) {
		framebuffer[index] = glm::vec3(0, 0, 0);

		for (int i = 0; i < numLights; ++i) {
			float lambert = glm::clamp(glm::dot(fragmentBuffer[index].eyeNor, lights[i]), 0.f, 1.f);

			framebuffer[index] += fragmentBuffer[index].color * lambert;
		}

		// TODO: add your fragment shader code here

	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
	originalWidth = w;
	originalHeight = h;
	width = antialiasing * originalWidth;
	height = antialiasing * originalHeight;

	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	hipFree(dev_framebuffer);
	hipMalloc(&dev_framebuffer, width * height * sizeof(glm::vec3));
	hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(float));

	hipMalloc((void **)&mutex, width * height * sizeof(int));
	hipMemset(mutex, 0, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, float * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = 1.0;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {

	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {

			dev_dst[count * componentTypeByteSize * n
				+ offset * componentTypeByteSize
				+ j]

				=

				dev_src[byteOffset
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride)
				+ offset * componentTypeByteSize
				+ j];
		}
	}


}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {

	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	}
	else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode(
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
)
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);

									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();

					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
						});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}

	printf("Num primitives: %u\n", totalNumPrimitives);

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}


	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());

		//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__
void _vertexTransformAndAssembly(
	int numVertices,
	PrimitiveDevBufPointers primitive,
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal,
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space
		glm::vec3 &position = primitive.dev_position[vid];
		glm::vec3 &normal = primitive.dev_normal[vid];

		VertexOut &vertex = primitive.dev_verticesOut[vid];

		glm::vec4 projected = MVP * glm::vec4(position, 1);
		projected /= projected.w;

		vertex.pos = glm::vec4((projected.x + 1.f) * width * 0.5f, (1.f - projected.y) * height * 0.5f, (projected.z + 1.f) * 0.5f, 1.0);
		vertex.eyePos = glm::vec3(MV * glm::vec4(position, 1));
		vertex.eyeNor = glm::normalize(MV_normal * normal);

		// Give the vertex a random color or texture color
		if (primitive.dev_diffuseTex == NULL) {
			vertex.dev_diffuseTex = NULL;
			thrust::default_random_engine rng = thrust::default_random_engine(utilhash(vid + 11));
			thrust::uniform_real_distribution<float> u01(0, 1);
			vertex.color = glm::vec3(u01(rng), u01(rng), u01(rng));
		}
		else {
			vertex.dev_diffuseTex = primitive.dev_diffuseTex;
			vertex.texcoord0 = primitive.dev_texcoord0[vid];
			vertex.texWidth = primitive.diffuseTexWidth;
			vertex.texHeight = primitive.diffuseTexHeight;
		}

		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array

	}
}





static int curPrimitiveBeginId = 0;

__global__
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}


		// TODO: other primitive types (point, line)
	}

}

/**
* Rasterization
*/
/**
 * Rasterization kernel.
 */
__global__
void rasterizeKernel(int numPrimitives, int width, int height, Fragment *fragmentBuffer, Primitive* dev_primitives, float* depth, int* mutex) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx = x + (y * width);

	if (idx < numPrimitives) {
		Primitive &primitive = dev_primitives[idx];
		glm::vec3 tri[3] = { glm::vec3(primitive.v[0].pos), glm::vec3(primitive.v[1].pos), glm::vec3(primitive.v[2].pos) };

		AABB aabb = getAABBForTriangle(tri);
		aabb.min.x = glm::min((float)width - 1, glm::max(0.f, aabb.min.x));
		aabb.min.y = glm::min((float)height - 1, glm::max(0.f, aabb.min.y));
		aabb.max.x = glm::max(0.f, glm::min((float)width - 1, aabb.max.x));
		aabb.max.y = glm::max(0.f, glm::min((float)height - 1, aabb.max.y));

		for (int col = aabb.min.x; col <= aabb.max.x; ++col) {
			for (int row = aabb.min.y; row <= aabb.max.y; ++row) {
				glm::vec2 point = glm::vec2(col, row);
				int fragmentIndex = glm::min(width*height - 1, glm::max(0, col + (row * width)));

				glm::vec3 bary = calculateBarycentricCoordinate(tri, point);

				if (isBarycentricCoordInBounds(bary)) {
					bool isSet;
					do {
						isSet = (atomicCAS(&mutex[fragmentIndex], 0, 1) == 0);
						if (isSet) {
							Fragment &fragment = fragmentBuffer[fragmentIndex];

							// Only set this fragments attributes if closest depth
							float fragmentDepth = getZAtCoordinate(bary, tri);

							if (fragmentDepth < depth[fragmentIndex]) {
								depth[fragmentIndex] = fragmentDepth;

								// Perspective correct z
								glm::vec3 eyeTri[3] = { glm::vec3(primitive.v[0].eyePos), glm::vec3(primitive.v[1].eyePos), glm::vec3(primitive.v[2].eyePos) };
								float perspZ = perspectiveCorrectZ (eyeTri, bary);

								// Calculate normal
								glm::vec3 normals[3] = { primitive.v[0].eyeNor, primitive.v[1].eyeNor, primitive.v[2].eyeNor };
								fragment.eyeNor = glm::normalize( perspectiveCorrectInterpolation (eyeTri, perspZ, normals, bary));

								if (primitive.v[0].dev_diffuseTex) {
									glm::vec3 uv[3] = { glm::vec3(primitive.v[0].texcoord0, 0), glm::vec3(primitive.v[1].texcoord0, 0), glm::vec3(primitive.v[2].texcoord0, 0) };

									glm::vec2 final_uv = bary[0] * primitive.v[0].texcoord0 + bary[1] * primitive.v[1].texcoord0 + bary[2] * primitive.v[2].texcoord0;
									float u = final_uv.x * primitive.v[0].texWidth;
									float v = final_uv.y * primitive.v[0].texHeight;

									int uInt = glm::floor(u);
									int vInt = glm::floor(v);

									TextureData* texture = primitive.v[0].dev_diffuseTex;

									float u_fract = u - glm::floor(u);
									float v_fract = v - glm::floor(v);

									int col_00_offset = (uInt + (vInt * primitive.v[0].texWidth)) * 3;
									glm::vec3 col_00 = glm::vec3(texture[col_00_offset], texture[col_00_offset + 1], texture[col_00_offset + 2]);

									int col_10_offset = (uInt + 1 + (vInt * primitive.v[0].texWidth)) * 3;
									glm::vec3 col_10 = glm::vec3(texture[col_10_offset], texture[col_10_offset + 1], texture[col_10_offset + 2]);

									int col_01_offset = (uInt + ((vInt + 1) * primitive.v[0].texWidth)) * 3;
									glm::vec3 col_01 = glm::vec3(texture[col_01_offset], texture[col_01_offset + 1], texture[col_01_offset + 2]);

									int col_11_offset = (uInt + 1 + ((vInt + 1) * primitive.v[0].texWidth)) * 3;
									glm::vec3 col_11 = glm::vec3(texture[col_11_offset], texture[col_11_offset + 1], texture[col_11_offset + 2]);

									glm::vec3 col_interp1 = glm::mix(col_00, col_10, u_fract);
									glm::vec3 col_interp2 = glm::mix(col_01, col_11, u_fract);

									fragment.color = glm::mix(col_interp1, col_interp2, v_fract) / 255.f;
								}
								else {
									glm::vec3 colors[3] = { primitive.v[0].color, primitive.v[1].color, primitive.v[2].color };
									fragment.color = perspectiveCorrectInterpolation (eyeTri, perspZ, colors, bary);
								}
							}
						}
						if (isSet) {
							mutex[fragmentIndex] = 0;
						}
					} while (!isSet);
				}
			}
		}
	}
}


/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
	int sideLength2d = 8;
	dim3 blockSize2d(sideLength2d, sideLength2d);
	dim3 blockCount2d((width - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> > (p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices,
						curPrimitiveBeginId,
						dev_primitives,
						*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}

	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> > (width, height, dev_depth);

	// TODO: rasterize
	rasterizeKernel << <blockCount2d, blockSize2d >> > (totalNumPrimitives, width, height, dev_fragmentBuffer, dev_primitives, dev_depth, mutex);


	// Copy depthbuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> > (width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
	// Copy framebuffer into OpenGL buffer for OpenGL previewing
	sendImageToPBO << <blockCount2d, blockSize2d >> > (pbo, originalWidth, originalHeight, dev_framebuffer, antialiasing, width);
	checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

	// deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);


			//TODO: release other attributes and materials
		}
	}

	////////////

	hipFree(dev_primitives);
	dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

	hipFree(dev_framebuffer);
	dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

	hipFree(mutex);
	mutex = NULL;

	checkCUDAError("rasterize Free");
}