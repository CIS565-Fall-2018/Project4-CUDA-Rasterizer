#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <chrono>

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		glm::vec2 texcoord0;
		TextureData* dev_diffuseTex = NULL;
		int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;
		
        VertexAttributeTexcoord texcoord0;
		TextureData* dev_diffuseTex;
        int texWidth, texHeight;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test
static int * dev_mutex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__device__
glm::vec3 getTexColor(TextureData* tex, int width, float u, float v) 
{
    int index = u + v * width;
    return glm::vec3(tex[index * 3], tex[index * 3 + 1], tex[index * 3 + 2]) / 255.f;
}

// for more information on bilinear filtering:
// https://en.wikipedia.org/wiki/Bilinear_filtering
// used sample code from this source
__device__
glm::vec3 getBilinearFilteredPixelColor(Fragment &fragment)
{
    float u = fragment.texcoord0.x * fragment.texWidth - 0.5f;
    float v = fragment.texcoord0.y * fragment.texHeight - 0.5f;
    int x = glm::floor(u);
    int y = glm::floor(v);
    float u_ratio = u - x;
    float v_ratio = v - y;
    float u_opposite = 1.f - u_ratio;
    float v_opposite = 1.f - v_ratio;

    // retrieve texture data
    glm::vec3 texXY = getTexColor(fragment.dev_diffuseTex, fragment.texWidth, x, y);
    glm::vec3 texX1Y = getTexColor(fragment.dev_diffuseTex, fragment.texWidth, x + 1, y);
    glm::vec3 texXY1 = getTexColor(fragment.dev_diffuseTex, fragment.texWidth, x, y + 1);
    glm::vec3 texX1Y1 = getTexColor(fragment.dev_diffuseTex, fragment.texWidth, x + 1, y + 1);

    return (texXY * u_opposite + texX1Y * u_ratio) * v_opposite + 
           (texXY1 * u_opposite + texX1Y1 * u_ratio) * v_ratio;
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) 
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) 
    {
		// TODO: add your fragment shader code here
        Fragment fragment = fragmentBuffer[index];
 
        #if TEXTURE == 1
            if (fragment.dev_diffuseTex != NULL) 
            {
                #if BILINEAR == 1
                    fragment.color = getBilinearFilteredPixelColor(fragment);
                #else
                    int u = fragment.texcoord0.x * fragment.texWidth;
                    int v = fragment.texcoord0.y * fragment.texHeight;
                    fragment.color = getTexColor(fragment.dev_diffuseTex, fragment.texWidth, u, v);
                #endif
            }
        #endif
        
        framebuffer[index] = fragment.color;

        #if PRIMTYPE == 3
            framebuffer[index] *= glm::dot(fragment.eyeNor, glm::normalize(glm::vec3(1.0f) - fragmentBuffer[index].eyePos));
        #endif
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;

	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));

    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

    hipFree(dev_mutex);
    hipMalloc(&dev_mutex, width * height * sizeof(int));
    hipMemset(dev_mutex, 0, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) 
{
	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (vid < numVertices) 
    {
		// TODO: Apply vertex transformation here
        // Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
        glm::vec4 pos = MVP * glm::vec4(primitive.dev_position[vid], 1.0f);
        glm::vec3 eyePos = glm::vec3(MV * glm::vec4(primitive.dev_position[vid], 1.0f));
        glm::vec3 eyeNor = glm::normalize(MV_normal * primitive.dev_normal[vid]);
        // Then divide the pos by its w element to transform into NDC space
        if (pos.w != 0) pos /= pos.w;
        // Finally transform x and y to viewport space
        pos.x = 0.5f * (float)width * (pos.x + 1.f);
        pos.y = 0.5f * (float)height * (1.f - pos.y);
        // pos.z = 1.f / eyePos.z;

        // TODO: Apply vertex assembly here
        // Assemble all attribute arrays into the primitive array
        primitive.dev_verticesOut[vid].pos = pos;
        primitive.dev_verticesOut[vid].eyePos = eyePos;
        primitive.dev_verticesOut[vid].eyeNor = eyeNor;

        // retrieve texture data
        #if TEXTURE == 1
            primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];
            primitive.dev_verticesOut[vid].dev_diffuseTex = primitive.dev_diffuseTex;
            primitive.dev_verticesOut[vid].texWidth = primitive.diffuseTexWidth;
            primitive.dev_verticesOut[vid].texHeight = primitive.diffuseTexHeight;
        #endif
	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) 
    {
		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) 
        {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}

		// TODO: other primitive types (point, line)
	}
	
}

__device__
void _rasterizePoints(Fragment* dev_fragmentBuffer, Primitive& primitive, int width, int height)
{
    VertexOut v0 = primitive.v[0];
    VertexOut v1 = primitive.v[1];
    VertexOut v2 = primitive.v[2];
    glm::vec3 triangle[3] = { glm::vec3(v0.pos),glm::vec3(v1.pos),glm::vec3(v2.pos) };

    int x, y;
    for (int vertIdx = 0; vertIdx < 3; ++vertIdx) 
    {
        x = triangle[vertIdx].x; y = triangle[vertIdx].y;
        int fragmentId = x + y * width;
        if ( (x >= 0 && x <= width - 1) && (y >= 0 && y <= height - 1) )
        {
            dev_fragmentBuffer[fragmentId].color = glm::vec3(1.f);
        }
    }
}

__device__
void _rasterizeLines(Fragment* dev_fragmentBuffer, Primitive& primitive, const int *indicies, int width, int height)
{
    VertexOut v0 = primitive.v[0];
    VertexOut v1 = primitive.v[1];
    VertexOut v2 = primitive.v[2];
    glm::vec3 triangle[3] = { glm::vec3(v0.pos),glm::vec3(v1.pos),glm::vec3(v2.pos) };

    int x1, x2, y1, y2, dx, dy, y, fragmentId;
    for (int index = 0; index < 6; index += 2) 
    {
        x1 = triangle[indicies[index]].x;    
        y1 = triangle[indicies[index]].y;
        x2 = triangle[indicies[index + 1]].x;  
        y2 = triangle[indicies[index + 1]].y;
        dx = x2 - x1;                   
        dy = y2 - y1;
        for (int x = x1; x <= x2; x++) 
        {
            y = y1 + dy * (x - x1) / dx;
            fragmentId = x + y * width;
            if ( (x >= 0 && x <= width - 1) && (y >= 0 && y <= height - 1) )
            {
                dev_fragmentBuffer[fragmentId].color = glm::vec3(1.f);
            }
        }
    }
}

__device__
void _rasterizeTriangles(Fragment* dev_fragmentBuffer, Primitive& primitive, int* dev_depth, int* dev_mutex, int width, int height)
{
    VertexOut v0 = primitive.v[0];
    VertexOut v1 = primitive.v[1];
    VertexOut v2 = primitive.v[2];
    glm::vec3 triangle[3] = { glm::vec3(v0.pos),glm::vec3(v1.pos),glm::vec3(v2.pos) };

    // find the min and max of triangle bounding box
    AABB bBox = getAABBForTriangle(triangle);
    const int minX = glm::min(glm::max((int)bBox.min.x, 0), width - 1);
    const int minY = glm::min(glm::max((int)bBox.min.y, 0), height - 1);
    const int maxX = glm::min(glm::max((int)bBox.max.x, 0), width - 1);
    const int maxY = glm::min(glm::max((int)bBox.max.y, 0), height - 1);

    for (int x = minX; x <= maxX; x++)
    {
        for (int y = minY; y <= maxY; y++)
        {
            glm::vec3 barycentricCoord = calculateBarycentricCoordinate(triangle, glm::vec2(x, y));
            if (isBarycentricCoordInBounds(barycentricCoord))
            {
                Fragment fragment;
                fragment.eyePos = v0.eyePos * barycentricCoord.x + v1.eyePos * barycentricCoord.y + v2.eyePos * barycentricCoord.z;
                fragment.eyeNor = v0.eyeNor * barycentricCoord.x + v1.eyeNor * barycentricCoord.y + v2.eyeNor * barycentricCoord.z;
                // use texture color
                #if TEXTURE == 1
                    fragment.dev_diffuseTex = v0.dev_diffuseTex;
                    fragment.texWidth = v0.texWidth;
                    fragment.texHeight = v0.texHeight;
                    // perspective correct texture coordinates
                    #if PERSPECTIVE == 1
                        const float zCoord = 1.f / (barycentricCoord.x / v0.eyePos.z
                                                  + barycentricCoord.y / v1.eyePos.z
                                                  + barycentricCoord.z / v2.eyePos.z);
                        fragment.texcoord0 = zCoord * (barycentricCoord.x * (v0.texcoord0 / v0.eyePos.z)
                                                     + barycentricCoord.y * (v1.texcoord0 / v1.eyePos.z)
                                                     + barycentricCoord.z * (v2.texcoord0 / v2.eyePos.z));
                    // no perspective correct
                    #else
                        fragment.texcoord0 = barycentricCoord.x * v0.texcoord0 + barycentricCoord.y * v1.texcoord0 + barycentricCoord.z * v2.texcoord0;
                    #endif
                // do not use texture color
                #else
                    fragment.dev_diffuseTex = NULL;
                    // default use vertex normal as color
                    fragment.color = fragment.eyeNor;
                #endif

                const int fragIndex = x + (y * width);
                bool isSet;
                do
                {
                    isSet = (atomicCAS(&dev_mutex[fragIndex], 0, 1) == 0);
                    if (isSet)
                    {
                        int depth = -getZAtCoordinate(barycentricCoord, triangle) * INT_MAX;
                        if (depth < dev_depth[fragIndex])
                        {
                            dev_depth[fragIndex] = depth;
                            dev_fragmentBuffer[fragIndex] = fragment;
                        }

                        //reset mutex
                        dev_mutex[fragIndex] = 0;

                    }

                } while (!isSet);

            }
        }
    }
}

__global__
void _rasterize(int totalNumPrimitives, Primitive* dev_primitives,
    Fragment* dev_fragmentBuffer, int* dev_depth,
    int * dev_mutex, int width, int height)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index > totalNumPrimitives) return;

    // get the triangle vertices
    Primitive primitive = dev_primitives[index];
 
    #if PRIMTYPE == 1
        _rasterizePoints(dev_fragmentBuffer, primitive, width, height);
    #elif PRIMTYPE == 2
        const int indices[] = { 0, 1, 1, 2, 2, 0 };
        _rasterizeLines(dev_fragmentBuffer, primitive, indices, width, height);
    #elif PRIMTYPE == 3
        _rasterizeTriangles(dev_fragmentBuffer, primitive, dev_depth, dev_mutex, width, height);
    #endif
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) 
{
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

        #if TIMER
            using time_point_t = std::chrono::high_resolution_clock::time_point;
            time_point_t start_time = std::chrono::high_resolution_clock::now();
        #endif

		for (; it != itEnd; ++it) 
        {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}
		checkCUDAError("Vertex Processing and Primitive Assembly");

        #if TIMER
            hipDeviceSynchronize();
            time_point_t end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> dur = end_time - start_time;
            float elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
            std::cout << std::endl;
            std::cout << "Vertex Processing and Primitive Assembly: " << elapsed_time << " milliseconds." << std::endl;
        #endif

	}

	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	

    #if TIMER
        using time_point_t = std::chrono::high_resolution_clock::time_point;
        time_point_t start_time = std::chrono::high_resolution_clock::now();
    #endif

	// TODO: rasterize
    dim3 numThreadsPerBlock(128);
    dim3 numBlocksForPrimitives = (totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x;
    _rasterize << <numBlocksForPrimitives, numThreadsPerBlock >> > (totalNumPrimitives, dev_primitives, dev_fragmentBuffer, dev_depth, dev_mutex, width, height);

    #if TIMER
        hipDeviceSynchronize();
        time_point_t end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> dur = end_time - start_time;
        float elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
        std::cout << "Rasterization: " << elapsed_time << " milliseconds." << std::endl;
    #endif

    #if TIMER
        start_time = std::chrono::high_resolution_clock::now();
    #endif

    // Copy depthbuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");

    #if TIMER
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        dur = end_time - start_time;
        elapsed_time = static_cast<decltype(elapsed_time)>(dur.count());
        std::cout << "Fragment Shader: " << elapsed_time << " milliseconds." << std::endl;
    #endif

    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    hipFree(dev_mutex);
    dev_mutex = NULL;

    checkCUDAError("rasterize Free");
}
